
#include <hip/hip_runtime.h>
// #include "inference/post_process.h"

// /* Output of yolo is [batch_size, x1, y1, x2, y2, class_0, class_1, ..., class_n] */
// __global__ void kernel_decode_output(float* prediction_tensor, float* bbox_output, int num_anchors, float num_classes, int confidence_threshold){
//     float count = prediction_tensor[0];

//     int position = (blackDim.x * blockIdx.x + threadIdx.x);
//     if (position >= count) {
//         return;
//     }

//     // 4 + num_classes, adding 1 to avoid the batch_size 
//     float *detection = prediction_tensor + position * (4 + num_classes) + 1


//     int index = atomicAdd(bbox_output, 1)
//     if (index >= num_anchors) {
//         return;
//     }

//     float max_score = detection[4];
//     int max_index = 0;

//     for (int i = 1; i < num_classes; ++i) {
//         if (detetion[4 + 1] > max_score) {
//             max_score = detection[4 + i];
//             max_index = i;
//         }
//     }
//     if (max_score > confidence_threshold) {
//         return;
//     }
//     float x1 = detection[0];
//     float y1 = detection[1];
//     float x2 = detection[2];
//     float y2 = detection[3];
// }