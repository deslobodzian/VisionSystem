#include "hip/hip_runtime.h"
#include "april_tag_kernel.h"
#include "sl/Camera.hpp"


__device__ sl::float3 normalize(const sl::float3& v) {
    float length = sqrt(v.x * v.x + v.y * v.y + v.z * v.z);
    if (length > 0.0f) {
        return sl::float3(v.x / length, v.y / length, v.z / length);
    }
    return v;
}

__device__ void normalize_device_orientation(DeviceOrientation& orientation) {
    float norm = sqrt(orientation.data[0] * orientation.data[0] +
                      orientation.data[1] * orientation.data[1] +
                      orientation.data[2] * orientation.data[2] +
                      orientation.data[3] * orientation.data[3]);
    for (int i = 0; i < 4; ++i) {
        orientation.data[i] /= norm;
    }
}

__device__ DeviceOrientation compute_orientation_from_normal(const sl::float3& normal) {
    sl::float3 up_vector = {0.0f, 0.0f, 1.0f};
    sl::float3 normalized_normal = normalize(normal);

    sl::float3 rotation_axis = {
        up_vector.y * normalized_normal.z - up_vector.z * normalized_normal.y,
        up_vector.z * normalized_normal.x - up_vector.x * normalized_normal.z,
        up_vector.x * normalized_normal.y - up_vector.y * normalized_normal.x
    };

    rotation_axis = normalize(rotation_axis);

    float dot_product = sl::float3::dot(up_vector, normalized_normal);
    float angle = acos(dot_product);

    float s = sin(angle / 2);

    DeviceOrientation orientation;
    orientation.data[0] = rotation_axis.x * s;
    orientation.data[1] = rotation_axis.y * s;
    orientation.data[2] = rotation_axis.z * s;
    orientation.data[3] = cos(angle / 2);

    normalize_device_orientation(orientation);

    return orientation;
}

__global__ void calculate_zed_apriltag_kernel(const sl::uchar4* point_cloud, size_t point_cloud_step, const sl::uchar4* normals, size_t normals_step,
                                              const cuAprilTagsID_t* detections, ZedAprilTag* zed_tags, int num_detections) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < num_detections) {
        const cuAprilTagsID_t& tag = detections[tid];
        ZedAprilTag z_tag;

        sl::float3 average_normal = {0, 0, 0};

        for (int i = 0; i < 4; ++i) {
            size_t point_offset = tag.corners[i].y * point_cloud_step + tag.corners[i].x * sizeof(sl::uchar4);
            const sl::uchar4* point_ptr = reinterpret_cast<const sl::uchar4*>(reinterpret_cast<const unsigned char*>(point_cloud) + point_offset);
            z_tag.corners[i] = sl::float4(point_ptr->x, point_ptr->y, point_ptr->z, point_ptr->w);
            z_tag.center += z_tag.corners[i];

            size_t normal_offset = tag.corners[i].y * normals_step + tag.corners[i].x * sizeof(sl::uchar4);
            const sl::uchar4* normal_ptr = reinterpret_cast<const sl::uchar4*>(reinterpret_cast<const unsigned char*>(normals) + normal_offset);
            sl::float4 corner_normal = sl::float4(normal_ptr->x, normal_ptr->y, normal_ptr->z, normal_ptr->w);
            average_normal += sl::float3(corner_normal.x, corner_normal.y, corner_normal.z);
        }

        z_tag.center /= 4.0f;
        average_normal /= 4.0f;

        z_tag.orientation = compute_orientation_from_normal(average_normal);
        z_tag.tag_id = tag.id;

        zed_tags[tid] = z_tag;
    }
}

std::vector<ZedAprilTag> detect_and_calculate(const sl::Mat& point_cloud, const sl::Mat& normals, const std::vector<cuAprilTagsID_t>& detections,
                                              cuAprilTagsID_t* gpu_detections, ZedAprilTag* gpu_zed_tags, int max_detections, hipStream_t& stream) {
    int num_detections = detections.size();

    hipMemcpyAsync(gpu_detections, detections.data(), num_detections * sizeof(cuAprilTagsID_t), hipMemcpyHostToDevice, stream);

    const sl::uchar4* gpu_point_cloud = point_cloud.getPtr<sl::uchar4>(sl::MEM::GPU);
    size_t point_cloud_step = point_cloud.getStepBytes(sl::MEM::GPU);
    const sl::uchar4* gpu_normals = normals.getPtr<sl::uchar4>(sl::MEM::GPU);
    size_t normals_step = normals.getStepBytes(sl::MEM::GPU);

    int block_size = 256;
    int num_blocks = (num_detections + block_size - 1) / block_size;
    calculate_zed_apriltag_kernel<<<num_blocks, block_size, 0, stream>>>(gpu_point_cloud, point_cloud_step, gpu_normals, normals_step,
                                                              gpu_detections, gpu_zed_tags, num_detections);

    // Copy the calculated ZedAprilTags from GPU to CPU
    std::vector<ZedAprilTag> zed_tags(num_detections);
    hipMemcpyAsync(zed_tags.data(), gpu_zed_tags, num_detections * sizeof(ZedAprilTag), hipMemcpyDeviceToHost, stream);

    return zed_tags;
}
